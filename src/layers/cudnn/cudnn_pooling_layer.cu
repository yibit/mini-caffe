
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN

#include "./cudnn_pooling_layer.hpp"

namespace caffe {

void CuDNNPoolingLayer::Forward_gpu(const vector<Blob*>& bottom,
                                    const vector<Blob*>& top) {
  const real_t* bottom_data = bottom[0]->gpu_data();
  real_t* top_data = top[0]->mutable_gpu_data();
  CUDNN_CHECK(cudnnPoolingForward(handle_, pooling_desc_,
              cudnn::dataType<real_t>::one,
              bottom_desc_, bottom_data,
              cudnn::dataType<real_t>::zero,
              top_desc_, top_data));
}

}  // namespace caffe

#endif  // USE_CUDNN
